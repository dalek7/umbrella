/*
  multiplication table using CUDA
  code downloaded from http://blog.daum.net/heoly/7
  $ nvcc -o multiplication multiplication.cu
*/
#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 5 // upto 5+1 dan
#define THREAD_SIZE 9

// Device code
__global__ void test(int *result)
{
    int tidx, bidx;
    tidx = threadIdx.x; //x-coordinate of thread
    bidx = blockIdx.x; //x-coordinate of block

    result[THREAD_SIZE * bidx + tidx] = (bidx + 2) * (tidx + 1);
}

// Host code
int main()
{
    int *host_Result; //Save result data of host
    int *device_Result; //Save result data of device

    int i=0, j=0;

    //Allocate host memory
    host_Result = (int *)malloc( BLOCK_SIZE * THREAD_SIZE * sizeof(int) );

    //Allocate device memory
    hipMalloc( (void**) &device_Result, sizeof(int) * BLOCK_SIZE * THREAD_SIZE);

    //Function name <<BLOCK_SIZE, THREAD_SIZE>>> parameters
    test <<<BLOCK_SIZE, THREAD_SIZE>>>(device_Result); //Execute Device code

    //Copy device result to host result
    hipMemcpy( host_Result, device_Result, sizeof(int) * BLOCK_SIZE * THREAD_SIZE, hipMemcpyDeviceToHost );


    //Print result
    for(j=0; j<BLOCK_SIZE; j++)
    {
        printf("%3d step\n", (j + 2));
        for(i=0; i<THREAD_SIZE; i++)
        {
            printf("%3d X %3d = %3d\n", j+2, i+1, host_Result[j * THREAD_SIZE + i]);
        }
        printf("\n");
    }
    free(host_Result); //Free host memory
    hipFree(device_Result); //Free device memory

    return 1;
}
